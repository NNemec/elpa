#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_complex.h>
#include "config-f90.h"

// Reset a reduction block
// Limitation: the thread-block size must be a divider of the reduction block's size
#ifdef DOUBLE_PRECISION_REAL
__device__ void reset_shared_block_c ( double * s_block, int b_size)
#else
__device__ void reset_shared_block_c ( float * s_block, int b_size)
#endif
{
    int i, t_idx, s_chunk ;
    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1) ; i < (t_idx * s_chunk); i++)
        s_block[i] = 0.0 ;
    __syncthreads();
}

// Reset 2 reduction blocks without an explicit synchronization at the end
// Limitation: : the thread-block size must be a divider of the reduction block's size
#ifdef DOUBLE_PRECISION_REAL
__device__ void reset_shared_block_pair_c( double *s_block_1, double *s_block_2, int b_size)
#else
__device__ void reset_shared_block_pair_c( float *s_block_1, float *s_block_2, int b_size)
#endif
{
    int i, t_idx, s_chunk;

    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1); i < (t_idx * s_chunk); i++)
    {    s_block_1[i] = 0.0 ;
        s_block_2[i] = 0.0 ;
    }
}
// Reset a reduction block
// Limitation: the thread-block size must be a divider of the reduction block's size
#ifdef DOUBLE_PRECISION_COMPLEX
__device__ void reset_shared_block_c_complex ( hipDoubleComplex * s_block, int b_size)
#else
__device__ void reset_shared_block_c_complex ( hipFloatComplex * s_block, int b_size)
#endif
{
    int i, t_idx, s_chunk ;
    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1) ; i < (t_idx * s_chunk); i++)
       { s_block[i].x = 0.0 ;
        s_block[i].y = 0.0 ;}
    __syncthreads();
}

// Reset 2 reduction blocks without an explicit synchronization at the end
// Limitation: : the thread-block size must be a divider of the reduction block's size
#ifdef DOUBLE_PRECISION_COMPLEX
__device__ void reset_shared_block_pair_c_complex( hipDoubleComplex *s_block_1, hipDoubleComplex *s_block_2, int b_size)
#else
__device__ void reset_shared_block_pair_c_complex( hipFloatComplex *s_block_1, hipFloatComplex *s_block_2, int b_size)
#endif
{
    int i, t_idx, s_chunk;

    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1); i < (t_idx * s_chunk); i++)
    {    s_block_1[i].x = 0.0 ;
        s_block_2[i].x= 0.0 ;
        s_block_1[i].y = 0.0 ;
        s_block_2[i].y= 0.0 ;
    }
}
#ifdef DOUBLE_PRECISION_COMPLEX
__device__ void warp_reduce_complex( hipDoubleComplex *s_block)
#else
__device__ void warp_reduce_complex( hipFloatComplex *s_block)
#endif
{
    int t_idx ;
    t_idx = threadIdx.x;
    __syncthreads();

	if (t_idx < 32)
        {

        s_block[t_idx] = hipCadd(hipCadd(s_block[t_idx],s_block[t_idx + 32]) , hipCadd( s_block[t_idx + 64], s_block[t_idx + 96]) );
        if (t_idx < 8)
        {
        s_block[t_idx] = hipCadd(hipCadd(s_block[t_idx],s_block[t_idx + 8] ) , hipCadd( s_block[t_idx + 16] , s_block[t_idx + 24] ) );

        }
        if (t_idx < 4)
        {
        s_block[t_idx] = hipCadd(s_block[t_idx] , s_block[t_idx + 4]) ;
        }
        if (t_idx < 1)
        {
        s_block[t_idx] = hipCadd(hipCadd(s_block[t_idx],s_block[t_idx + 1] ) , hipCadd( s_block[t_idx +2] , s_block[t_idx + 3] ) );
        }
        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void my_pack_c_kernel_complex(const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* src, hipDoubleComplex* dst)
#else
__global__ void my_pack_c_kernel_complex(const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipFloatComplex* src, hipFloatComplex* dst)
#endif
{
    int b_id, t_id ;
    int dst_ind ;
    b_id = blockIdx.y;
    t_id = threadIdx.x;

    dst_ind = b_id * stripe_width + t_id;
    if (dst_ind < max_idx)
    {
        // dimension of dst - lnev, nblk
        // dimension of src - stripe_width,a_dim2,stripe_count
	dst[dst_ind + (l_nev*blockIdx.x)].x = src[t_id + (stripe_width*(n_offset + blockIdx.x)) + ( b_id *stripe_width*a_dim2)].x;
        dst[dst_ind + (l_nev*blockIdx.x)].y = src[t_id + (stripe_width*(n_offset + blockIdx.x)) + ( b_id *stripe_width*a_dim2)].y;
     }

}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void  my_unpack_c_kernel_complex( const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* src, hipDoubleComplex* dst)
#else
__global__ void  my_unpack_c_kernel_complex( const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipFloatComplex* src, hipFloatComplex* dst)
#endif
{
    int b_id, t_id ;
    int src_ind;

    b_id = blockIdx.y;
    t_id = threadIdx.x;

    src_ind = b_id * stripe_width + t_id;
    if (src_ind < max_idx)
{	dst[ t_id + ((n_offset + blockIdx.x) * stripe_width) + (b_id * stripe_width * a_dim2 )].x = src[ src_ind  + (blockIdx.x) *l_nev ].x;
	dst[ t_id + ((n_offset + blockIdx.x) * stripe_width) + (b_id * stripe_width * a_dim2 )].y = src[ src_ind  + (blockIdx.x) *l_nev ].y;
}
}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void extract_hh_tau_c_kernel_complex(hipDoubleComplex* hh, hipDoubleComplex* hh_tau, const int nbw, const int n, int val)
#else
__global__ void extract_hh_tau_c_kernel_complex(hipFloatComplex* hh, hipFloatComplex* hh_tau, const int nbw, const int n, int val)
#endif
{
    int h_idx ;

    h_idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    if (h_idx < n)
    {
        //dimension of hh - (nbw, max_blk_size)
        //dimension of hh_tau - max_blk_size
        hh_tau[h_idx] = hh[h_idx * nbw] ;
        //  Replace the first element in the HH reflector with 1.0 or 0.0
        if( val == 0)
        {
         hh[(h_idx * nbw)].x = 1.0;
	 hh[h_idx *nbw].y= 0.0;
        }
        else
        {
        hh[(h_idx * nbw)].x = 0.0;
	hh[h_idx*nbw].y =0.0;
        }
     }
}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void  compute_hh_dotp_c_kernel_complex(hipDoubleComplex* hh, hipDoubleComplex* v_dot, const int nbw, const int n)
{
   __shared__ hipDoubleComplex hh_s[128] ;
#else
__global__ void  compute_hh_dotp_c_kernel_complex(hipFloatComplex* hh, hipFloatComplex* v_dot, const int nbw, const int n)
{
   __shared__ hipFloatComplex hh_s[128] ;
#endif

    int t_idx, v_idx;

    //  The vector index (v_idx) identifies the pair of HH reflectors from which the dot product is computed
    v_idx = blockIdx.x  ;

    //  The thread index indicates the position within the two HH reflectors
    t_idx = threadIdx.x ;

    if (t_idx  > 0)
    {

       hh_s[t_idx] = hipCmul(hipConj(hh[t_idx + v_idx * nbw]),   hh[ (t_idx - 1) +  (v_idx +1)* nbw]) ;
    }
    else
    {
        hh_s[t_idx].x = 0.0 ;
        hh_s[t_idx].y = 0.0;
    }

  //  Compute the dot product using a fast reduction
     warp_reduce_complex(hh_s);
     __syncthreads();

      if(t_idx == 0)
       {
	v_dot[v_idx] = hh_s[0] ;
	}

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_my_pack_c_kernel_complex(const int row_count, const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* a_dev, hipDoubleComplex* row_group_dev)
#else
extern "C" void launch_my_pack_c_kernel_complex(const int row_count, const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipFloatComplex* a_dev, hipFloatComplex* row_group_dev)
#endif
{

        dim3  grid_size;
        grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to mypack kernel: %s, %d\n",hipGetErrorString(err), err);
        my_pack_c_kernel_complex<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, a_dev, row_group_dev);
        err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n my pack_kernel failed  %s \n",hipGetErrorString(err) );
        }
}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_compute_hh_dotp_c_kernel_complex(hipDoubleComplex* bcast_buffer_dev, hipDoubleComplex* hh_dot_dev,const int nbw,const int n)
#else
extern "C" void launch_compute_hh_dotp_c_kernel_complex(hipFloatComplex* bcast_buffer_dev, hipFloatComplex* hh_dot_dev,const int nbw,const int n)
#endif
{
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to compute_hh kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_hh_dotp_c_kernel_complex<<< n-1, nbw >>>(bcast_buffer_dev, hh_dot_dev, nbw, n);

        err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n compute _kernel failed  %s \n",hipGetErrorString(err) );
        }
}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_extract_hh_tau_c_kernel_complex(hipDoubleComplex* bcast_buffer_dev, hipDoubleComplex* hh_tau_dev, const int nbw, const int n , const int is_zero)
#else
extern "C" void launch_extract_hh_tau_c_kernel_complex(hipFloatComplex* bcast_buffer_dev, hipFloatComplex* hh_tau_dev, const int nbw, const int n , const int is_zero)
#endif
{
        int grid_size;
        grid_size = 1 + (n - 1) / 256;
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to extract kernel: %s, %d\n",hipGetErrorString(err), err);
        extract_hh_tau_c_kernel_complex<<<grid_size,256>>>(bcast_buffer_dev,hh_tau_dev, nbw, n, is_zero);
        err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n  extract _kernel failed  %s \n",hipGetErrorString(err) );
        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_my_unpack_c_kernel_complex( const int row_count, const int n_offset, const int max_idx, const int stripe_width,const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* row_group_dev, hipDoubleComplex* a_dev)
#else 
extern "C" void launch_my_unpack_c_kernel_complex( const int row_count, const int n_offset, const int max_idx, const int stripe_width,const int a_dim2, const int stripe_count, const int l_nev, hipFloatComplex* row_group_dev, hipFloatComplex* a_dev)
#endif
{

        dim3  grid_size;
        grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to unpack kernel: %s, %d\n",hipGetErrorString(err), err);
        my_unpack_c_kernel_complex<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, row_group_dev , a_dev);
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n  my_unpack_c_kernel failed  %s \n",hipGetErrorString(err) );
        }
}

#ifdef DOUBLE_PRECISION_REAL
__device__ void warp_reduce_c( double *s_block)
#else
__device__ void warp_reduce_c( float *s_block)
#endif
{
    int t_idx ;
    t_idx = threadIdx.x;
    __syncthreads();

        if (t_idx < 32)
	{
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 32] + s_block[t_idx + 64] + s_block[t_idx + 96] ;
        if (t_idx < 8)
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 8] + s_block[t_idx + 16] + s_block[t_idx + 24];
        if (t_idx < 4)
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 4];
        if (t_idx < 1)
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 1] + s_block[t_idx + 2] + s_block[t_idx + 3];
	}
}

#ifdef DOUBLE_PRECISION_REAL
__global__ void my_pack_c_kernel(const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, double* src, double* dst)
#else
__global__ void my_pack_c_kernel(const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, float* src, float* dst)
#endif
{
    int b_id, t_id ;
    int dst_ind ;
    b_id = blockIdx.y;
    t_id = threadIdx.x;

    dst_ind = b_id * stripe_width + t_id;
    if (dst_ind < max_idx)
    {
	// dimension of dst - lnev, nblk
	// dimension of src - stripe_width,a_dim2,stripe_count
    	*(dst + dst_ind + (l_nev*blockIdx.x) ) = *(src + t_id + (stripe_width*(n_offset + blockIdx.x)) + ( b_id *stripe_width*a_dim2 ));
     }

}

#ifdef DOUBLE_PRECISION_REAL
__global__ void  my_unpack_c_kernel( const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, double* src, double* dst)
#else
__global__ void  my_unpack_c_kernel( const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, float* src, float* dst)
#endif
{
    int b_id, t_id ;
    int src_ind;

    b_id = blockIdx.y;
    t_id = threadIdx.x;

    src_ind = b_id * stripe_width + t_id;
    if (src_ind < max_idx)
	*(dst + (t_id + ((n_offset + blockIdx.x) * stripe_width) + (b_id * stripe_width * a_dim2 ))) = *(src + src_ind  + (blockIdx.x) *l_nev );

}
#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void compute_kernel_reduce( hipDoubleComplex* a_dev, int lda , int n ,int nbw ,  hipDoubleComplex *h1_dev )
#else
__global__ void compute_kernel_reduce( hipFloatComplex* a_dev, int lda , int n ,int nbw ,  hipFloatComplex *h1_dev )
#endif
{
    int  t_id ;
    int st_ind;

    t_id = threadIdx.x;

    st_ind = (t_id*(t_id+1))/2;
    if(t_id< n)
    {
	for(int i =0;i<=t_id;i++)
        {
         h1_dev[st_ind + i] = a_dev[t_id *lda + i ] ;
	}
    }
    __syncthreads();


}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void compute_kernel_reduce_1( hipDoubleComplex* a_dev, int lda , int n, hipDoubleComplex *h1_dev )
#else
__global__ void compute_kernel_reduce_1( hipFloatComplex* a_dev, int lda , int n, hipFloatComplex *h1_dev )
#endif
{
    int  t_id ;
    int st_ind;

    t_id = threadIdx.x;

    st_ind = (t_id*(t_id+1))/2;
    if(t_id< n)
    {
        for(int i =0;i<=t_id;i++)
         {
	  a_dev[t_id *lda + i ] = h1_dev[st_ind + i];
	  a_dev[ (i-1)*lda + t_id ] = hipConj(a_dev[ t_id *lda + i-1]) ;
	}
    }
    __syncthreads();


}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void  dot_product_c_kernel( hipDoubleComplex* hs_dev, hipDoubleComplex* hv_new_dev, hipDoubleComplex tau_new_dev, hipDoubleComplex*  x_dev, hipDoubleComplex *h_dev, hipDoubleComplex *hv_dev, int nr)
#else
__global__ void  dot_product_c_kernel( hipFloatComplex* hs_dev, hipFloatComplex* hv_new_dev, hipFloatComplex tau_new_dev, hipFloatComplex*  x_dev, hipFloatComplex *h_dev, hipFloatComplex *hv_dev, int nr)
#endif
{
    int t_id ;

#ifdef DOUBLE_PRECISION_COMPLEX
    __shared__ hipDoubleComplex x_dev_temp[128];
    __shared__ hipDoubleComplex x_val;
#else
    __shared__ hipFloatComplex x_dev_temp[128];
    __shared__ hipFloatComplex x_val;
#endif
    //b_id = blockIdx.y;
    t_id = threadIdx.x;

    if(t_id<nr)
	 x_dev_temp[t_id] = hipCmul( hipConj(hs_dev[t_id]), hv_new_dev[t_id]) ;
    __syncthreads();

    if(t_id==0)
    {
        for(int i=1;i<nr;i++)
	x_dev_temp[t_id] = hipCadd(x_dev_temp[t_id],x_dev_temp[t_id +i]);
    }
    __syncthreads();
     if(t_id ==0)
    {
      x_val =  hipCmul(x_dev_temp[t_id], tau_new_dev);
      x_dev[0] = x_val;
    }
	__syncthreads();
}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void  dot_product_c_kernel_1(   hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,  hipDoubleComplex*  hv_new_dev, hipDoubleComplex*  x_dev, hipDoubleComplex *h_dev, hipDoubleComplex *hv_dev,  int nb, int nr , int ns )
#else
__global__ void  dot_product_c_kernel_1(   hipFloatComplex*  ab_dev, hipFloatComplex *hs_dev,  hipFloatComplex*  hv_new_dev, hipFloatComplex*  x_dev, hipFloatComplex *h_dev, hipFloatComplex *hv_dev,  int nb, int nr , int ns )
#endif
{
    int t_id = threadIdx.x;
        int i;

    if((t_id>0 )&& (t_id < nb))
    {
	h_dev[t_id] = hipCsub(h_dev[t_id], hipCmul(x_dev[0],hv_dev[t_id]));
        for(i=0;i<nr;i++)
	{
	 ab_dev[ i+nb-t_id + (t_id+ns-1)*2*nb ] = hipCsub(hipCsub(ab_dev[ i+nb-t_id + (t_id+ns-1)*2*nb],hipCmul(hv_new_dev[i],hipConj(h_dev[t_id])) ),hipCmul(hs_dev[i], hipConj(hv_dev[t_id])));
 	}
    }

   __syncthreads();

}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void  double_hh_transform_kernel( hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev, hipDoubleComplex *hv_dev,  int nb,  int ns )
#else
__global__ void  double_hh_transform_kernel( hipFloatComplex*  ab_dev, hipFloatComplex *hs_dev, hipFloatComplex *hv_dev,  int nb,  int ns )
#endif
{
    int t_id = threadIdx.x;
    if((t_id>0 )&& (t_id < nb))
    {
         ab_dev[ nb-t_id + (t_id+ns-1)*2*nb ] = hipCsub(ab_dev[ nb-t_id + (t_id+ns-1)*2*nb],hipCmul(hs_dev[0], hipConj(hv_dev[t_id])));
    }

   __syncthreads();

}

#ifdef DOUBLE_PRECISION_COMPLEX
__global__ void  double_hh_transform_kernel_2( hipDoubleComplex*  ab_dev, hipDoubleComplex *hd_dev, hipDoubleComplex *hv_dev,  int nc,  int ns , int nb )
#else
__global__ void  double_hh_transform_kernel_2( hipFloatComplex*  ab_dev, hipFloatComplex *hd_dev, hipFloatComplex *hv_dev,  int nc,  int ns , int nb )
#endif
{
    int t_id = threadIdx.x;
    if(t_id < nc)
    {

         ab_dev[ t_id + (ns-1)*2*nb ] = hipCsub(hipCsub(ab_dev[ t_id + (ns-1)*2*nb],hipCmul(hd_dev[ t_id], hipConj(hv_dev[0]))) , hipCmul(hv_dev[ t_id], hipConj(hd_dev[0])));

    }

   __syncthreads();

}

#ifdef DOUBLE_PRECISION_REAL
__global__ void extract_hh_tau_c_kernel(double* hh, double* hh_tau, const int nbw, const int n, int val)
#else
__global__ void extract_hh_tau_c_kernel(float* hh, float* hh_tau, const int nbw, const int n, int val)
#endif
{
    int h_idx ;
    h_idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    if (h_idx < n)
    {
	//dimension of hh - (nbw, max_blk_size)
	//dimension of hh_tau - max_blk_size
        *(hh_tau + h_idx ) = *(hh +  (h_idx * nbw)) ;
        //  Replace the first element in the HH reflector with 1.0 or 0.0
	if( val == 0)
        *(hh + (h_idx * nbw)) = 1.0;
	else
	*(hh + (h_idx * nbw)) = 0.0;
     }
}

#ifdef DOUBLE_PRECISION_REAL
__global__ void  compute_hh_dotp_c_kernel(double* hh, double* v_dot, const int nbw, const int n)
{

   __shared__ double hh_s[128] ;
#else
__global__ void  compute_hh_dotp_c_kernel(float* hh, float* v_dot, const int nbw, const int n)
{

   __shared__ float hh_s[128] ;
#endif
    int t_idx, v_idx;

    //  The vector index (v_idx) identifies the pair of HH reflectors from which the dot product is computed
    v_idx = blockIdx.x  ;

    //  The thread index indicates the position within the two HH reflectors
    t_idx = threadIdx.x ;

//    //  The contents of the shared memory must be fully reset
//     reset_shared_block_c(hh_s, 128);

    //  Initialize the contents of the shared buffer (preparing for reduction)
    if (t_idx  > 0)
        *(hh_s + t_idx) = *(hh + t_idx + v_idx * nbw ) *  (*(hh + (t_idx - 1) +  (v_idx +1)* nbw)) ;
    else
        *(hh_s + t_idx) = 0.0 ;

     //  Compute the dot product using a fast reduction
     warp_reduce_c(hh_s);

      if(t_idx == 0)
      *(v_dot + v_idx) = *(hh_s) ;

}

#ifdef DOUBLE_PRECISION_REAL
extern "C" void launch_my_pack_c_kernel(const int row_count, const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, double* a_dev, double* row_group_dev)
#else
extern "C" void launch_my_pack_c_kernel(const int row_count, const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, float* a_dev, float* row_group_dev)
#endif
{

	dim3  grid_size;
        grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to mypack kernel: %s, %d\n",hipGetErrorString(err), err);

	my_pack_c_kernel<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, a_dev, row_group_dev);
	 err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n my pack_kernel failed  %s \n",hipGetErrorString(err) );
        }

}
#ifdef DOUBLE_PRECISION_REAL
extern "C" void launch_compute_hh_dotp_c_kernel(double* bcast_buffer_dev, double* hh_dot_dev,const int nbw,const int n)
#else
extern "C" void launch_compute_hh_dotp_c_kernel(float* bcast_buffer_dev, float* hh_dot_dev,const int nbw,const int n)
#endif
{
	hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to compute_hh kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_hh_dotp_c_kernel<<< n-1, nbw >>>(bcast_buffer_dev, hh_dot_dev, nbw, n);
	err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n compute _kernel failed  %s \n",hipGetErrorString(err) );
        }

}
#ifdef DOUBLE_PRECISION_REAL
extern "C" void launch_extract_hh_tau_c_kernel(double* bcast_buffer_dev, double* hh_tau_dev, const int nbw, const int n , const int is_zero)
#else
extern "C" void launch_extract_hh_tau_c_kernel(float* bcast_buffer_dev, float* hh_tau_dev, const int nbw, const int n , const int is_zero)
#endif
{
	int grid_size;
	grid_size = 1 + (n - 1) / 256;
	hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to extract kernel: %s, %d\n",hipGetErrorString(err), err);
	extract_hh_tau_c_kernel<<<grid_size,256>>>(bcast_buffer_dev,hh_tau_dev, nbw, n, is_zero);
	err = hipGetLastError();
	if ( err!= hipSuccess)
       	{
		printf("\n  extract _kernel failed  %s \n",hipGetErrorString(err) );
        }

}

#ifdef DOUBLE_PRECISION_REAL
extern "C" void launch_my_unpack_c_kernel( const int row_count, const int n_offset, const int max_idx, const int stripe_width,const int a_dim2, const int stripe_count, const int l_nev, double* row_group_dev, double* a_dev)
#else
extern "C" void launch_my_unpack_c_kernel( const int row_count, const int n_offset, const int max_idx, const int stripe_width,const int a_dim2, const int stripe_count, const int l_nev, float* row_group_dev, float* a_dev)
#endif
{

        dim3  grid_size;
	grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to unpack kernel: %s, %d\n",hipGetErrorString(err), err);
        my_unpack_c_kernel<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, row_group_dev , a_dev);
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
	    printf("\n  my_unpack_c_kernel failed  %s \n",hipGetErrorString(err) );
        }
}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_dot_product_kernel( hipDoubleComplex* hs_dev, hipDoubleComplex* hv_new_dev, hipDoubleComplex tau_new_dev, hipDoubleComplex*  x_dev, hipDoubleComplex*  h_dev ,hipDoubleComplex*  hv_dev,int nr )
#else
extern "C" void launch_dot_product_kernel( hipFloatComplex* hs_dev, hipFloatComplex* hv_new_dev, hipFloatComplex tau_new_dev, hipFloatComplex*  x_dev, hipFloatComplex*  h_dev ,hipFloatComplex*  hv_dev,int nr )
#endif
{

        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        dot_product_c_kernel<<<grid_size, nr>>>(hs_dev, hv_new_dev, tau_new_dev, x_dev, h_dev, hv_dev, nr );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_dot_product_kernel_1(  hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,  hipDoubleComplex*  hv_new_dev,hipDoubleComplex*  x_dev, hipDoubleComplex*  h_dev ,hipDoubleComplex*  hv_dev, int nb ,int nr , int ns)
#else
extern "C" void launch_dot_product_kernel_1(  hipFloatComplex*  ab_dev, hipFloatComplex *hs_dev,  hipFloatComplex*  hv_new_dev,hipFloatComplex*  x_dev, hipFloatComplex*  h_dev ,hipFloatComplex*  hv_dev, int nb ,int nr , int ns)
#endif
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        dot_product_c_kernel_1<<<grid_size, nb>>>( ab_dev, hs_dev, hv_new_dev, x_dev, h_dev, hv_dev, nb, nr, ns );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_dot_product_kernel_2(  hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,  hipDoubleComplex*  hv_dev,hipDoubleComplex*  hd_dev, int nb ,int nr , int ne)
#else
extern "C" void launch_dot_product_kernel_2(  hipFloatComplex*  ab_dev, hipFloatComplex *hs_dev,  hipFloatComplex*  hv_dev,hipFloatComplex*  hd_dev, int nb ,int nr , int ne)
#endif
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_double_hh_transform_1( hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,hipDoubleComplex*  hv_dev, int nb , int ns)
#else
extern "C" void launch_double_hh_transform_1( hipFloatComplex*  ab_dev, hipFloatComplex *hs_dev,hipFloatComplex*  hv_dev, int nb , int ns)
#endif
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_double_hh_transform kernel: %s, %d\n",hipGetErrorString(err), err);
        double_hh_transform_kernel<<<grid_size, nb>>>( ab_dev, hs_dev, hv_dev, nb,  ns );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_double_hh_transform_2( hipDoubleComplex*  ab_dev, hipDoubleComplex *hd_dev,hipDoubleComplex*  hv_dev, int nc , int ns , int nb )
#else
extern "C" void launch_double_hh_transform_2( hipFloatComplex*  ab_dev, hipFloatComplex *hd_dev,hipFloatComplex*  hv_dev, int nc , int ns , int nb )
#endif
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_double_hh_transform kernel: %s, %d\n",hipGetErrorString(err), err);
        double_hh_transform_kernel_2<<<grid_size, nc>>>( ab_dev, hd_dev, hv_dev, nc,  ns, nb );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_compute_kernel_reduce( hipDoubleComplex* a_dev, int lda, int n,int nbw, hipDoubleComplex* h_dev)
#else
extern "C" void launch_compute_kernel_reduce( hipFloatComplex* a_dev, int lda, int n,int nbw, hipFloatComplex* h_dev)
#endif
{

        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_kernel_reduce<<<grid_size,n>>>(a_dev, lda, n, nbw,h_dev);
	hipDeviceSynchronize();
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

#ifdef DOUBLE_PRECISION_COMPLEX
extern "C" void launch_compute_kernel_reduce_1( hipDoubleComplex* a_dev, int lda, int n , hipDoubleComplex* h_dev)
#else
extern "C" void launch_compute_kernel_reduce_1( hipFloatComplex* a_dev, int lda, int n , hipFloatComplex* h_dev)
#endif
{

        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_kernel_reduce_1<<<grid_size,n>>>(a_dev, lda, n, h_dev);
	hipDeviceSynchronize();
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}


extern "C" int cuda_MemcpyDeviceToDevice(int val)
{
      val = hipMemcpyDeviceToDevice;
      return val;
}
