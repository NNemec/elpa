#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <stdint.h>

#include "config-f90.h"

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

#ifdef DEBUG_CUDA
#define debugmessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)
#else
#define debugmessage(x, ...)
#endif

#ifdef WITH_GPU_VERSION
extern "C" {
  int cudaSetDeviceFromC(int n) {

    hipError_t cuerr = hipSetDevice(n);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipSetDevice: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaGetDeviceCountFromC(int *count) {

    hipError_t cuerr = hipGetDeviceCount(count);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaDeviceSynchronizeFromC() {

    hipError_t cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess) {
      errormessage("Error in hipGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }


  int cudaMallocFromC(intptr_t *a, size_t width_height) {

    hipError_t cuerr = hipMalloc((void **) a, width_height);
#ifdef DEBUG_CUDA
    printf("Malloc pointer address: %p \n", *a);
#endif
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMalloc: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaFreeFromC(intptr_t *a) {
#ifdef DEBUG_CUDA
    printf("Free pointer address: %p \n", a);
#endif
    hipError_t cuerr = hipFree(a);

    if (cuerr != hipSuccess) {
      errormessage("Error in hipFree: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemsetFromC(intptr_t *a, int value, size_t count) {

    hipError_t cuerr = hipMemset( a, value, count);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMemset: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpyFromC(intptr_t *dest, intptr_t *src, size_t count, int dir) {

    hipError_t cuerr = hipMemcpy( dest, src, count, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMemcpy: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpy2dFromC(intptr_t *dest, size_t dpitch, intptr_t *src, size_t spitch, size_t width, size_t height, int dir) {

    hipError_t cuerr = hipMemcpy2D( dest, dpitch, src, spitch, width, height, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMemcpy2d: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaMemcpyDeviceToDeviceFromC(void) {
      int val = hipMemcpyDeviceToDevice;
      return val;
  }
  int cudaMemcpyHostToDeviceFromC(void) {
      int val = hipMemcpyHostToDevice;
      return val;
  }
  int cudaMemcpyDeviceToHostFromC(void) {
      int val = hipMemcpyDeviceToHost;
      return val;
  }
  int cudaHostRegisterPortableFromC(void) {
      int val = hipHostRegisterPortable;
      return val;
  }
  int cudaHostRegisterMappedFromC(void) {
      int val = hipHostRegisterMapped;
      return val;
  }
}
#endif /* WITH_GPU_VERSION */
