#include <stdio.h>
#include <math.h>
#include <stdio.h>
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//
// --------------------------------------------------------------------------------------------------
//
// This file was written by A. Marek, MPCDF


#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <stdint.h>
#include <complex.h>
#include <hipblas.h>

#include "config-f90.h"

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

#ifdef DEBUG_CUDA
#define debugmessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)
#else
#define debugmessage(x, ...)
#endif

#ifdef WITH_GPU_VERSION
extern "C" {

  int cudaThreadSynchronizeFromC() {
    hipError_t cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess) {
      errormessage("Error in hipDeviceSynchronize: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }


  int cudaSetDeviceFromC(int n) {

    hipError_t cuerr = hipSetDevice(n);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipSetDevice: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaGetDeviceCountFromC(int *count) {

    hipError_t cuerr = hipGetDeviceCount(count);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaDeviceSynchronizeFromC() {

    hipError_t cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess) {
      errormessage("Error in hipGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }


  int cudaMallocFromC(intptr_t *a, size_t width_height) {

    hipError_t cuerr = hipMalloc((void **) a, width_height);
#ifdef DEBUG_CUDA
    printf("Malloc pointer address: %p \n", *a);
#endif
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMalloc: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaFreeFromC(intptr_t *a) {
#ifdef DEBUG_CUDA
    printf("Free pointer address: %p \n", a);
#endif
    hipError_t cuerr = hipFree(a);

    if (cuerr != hipSuccess) {
      errormessage("Error in hipFree: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemsetFromC(intptr_t *a, int value, size_t count) {

    hipError_t cuerr = hipMemset( a, value, count);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMemset: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpyFromC(intptr_t *dest, intptr_t *src, size_t count, int dir) {

    hipError_t cuerr = hipMemcpy( dest, src, count, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMemcpy: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpy2dFromC(intptr_t *dest, size_t dpitch, intptr_t *src, size_t spitch, size_t width, size_t height, int dir) {

    hipError_t cuerr = hipMemcpy2D( dest, dpitch, src, spitch, width, height, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMemcpy2d: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaMemcpyDeviceToDeviceFromC(void) {
      int val = hipMemcpyDeviceToDevice;
      return val;
  }
  int cudaMemcpyHostToDeviceFromC(void) {
      int val = hipMemcpyHostToDevice;
      return val;
  }
  int cudaMemcpyDeviceToHostFromC(void) {
      int val = hipMemcpyDeviceToHost;
      return val;
  }
  int cudaHostRegisterPortableFromC(void) {
      int val = hipHostRegisterPortable;
      return val;
  }
  int cudaHostRegisterMappedFromC(void) {
      int val = hipHostRegisterMapped;
      return val;
  }
  
  void cublasZgemv_elpa_wrapper (char trans, int m, int n, double complex alpha,
                               const double complex *A, int lda,  const double complex *x, int incx,
                               double complex beta, double complex *y, int incy) {    

    hipDoubleComplex alpha_casted = *((hipDoubleComplex*)(&alpha));
    hipDoubleComplex beta_casted = *((hipDoubleComplex*)(&beta));
    
    const hipDoubleComplex* A_casted = (const hipDoubleComplex*) A;
    const hipDoubleComplex* x_casted = (const hipDoubleComplex*) x;
    hipDoubleComplex* y_casted = (hipDoubleComplex*) y;
    
    hipblasZgemv(trans, m, n, alpha_casted, A_casted, lda, x_casted, incx, beta_casted, y_casted, incy);     
  }
  
  void cublasCgemv_elpa_wrapper (char trans, int m, int n, float complex alpha,
                               const float complex *A, int lda,  const float complex *x, int incx,
                               float complex beta, float complex *y, int incy) {    

    hipFloatComplex alpha_casted = *((hipFloatComplex*)(&alpha));
    hipFloatComplex beta_casted = *((hipFloatComplex*)(&beta));
    
    const hipFloatComplex* A_casted = (const hipFloatComplex*) A;
    const hipFloatComplex* x_casted = (const hipFloatComplex*) x;
    hipFloatComplex* y_casted = (hipFloatComplex*) y;
    
    hipblasCgemv(trans, m, n, alpha_casted, A_casted, lda, x_casted, incx, beta_casted, y_casted, incy);     
  }
  
  void cublasZgemm_elpa_wrapper (char transa, char transb, int m, int n, int k,
                               double complex alpha, const double complex *A, int lda,
                               const double complex *B, int ldb, double complex beta,
                               double complex *C, int ldc) {
    
    hipDoubleComplex alpha_casted = *((hipDoubleComplex*)(&alpha));
    hipDoubleComplex beta_casted = *((hipDoubleComplex*)(&beta));
    
    const hipDoubleComplex* A_casted = (const hipDoubleComplex*) A;
    const hipDoubleComplex* B_casted = (const hipDoubleComplex*) B;
    hipDoubleComplex* C_casted = (hipDoubleComplex*) C;
    
    hipblasZgemm(transa, transb, m, n, k, alpha_casted, A_casted, lda, B_casted, ldb, beta_casted, C_casted, ldc);
  }

  void cublasCgemm_elpa_wrapper (char transa, char transb, int m, int n, int k,
                               float complex alpha, const float complex *A, int lda,
                               const float complex *B, int ldb, float complex beta,
                               float complex *C, int ldc) {
    
    hipFloatComplex alpha_casted = *((hipFloatComplex*)(&alpha));
    hipFloatComplex beta_casted = *((hipFloatComplex*)(&beta));
    
    const hipFloatComplex* A_casted = (const hipFloatComplex*) A;
    const hipFloatComplex* B_casted = (const hipFloatComplex*) B;
    hipFloatComplex* C_casted = (hipFloatComplex*) C;
    
    hipblasCgemm(transa, transb, m, n, k, alpha_casted, A_casted, lda, B_casted, ldb, beta_casted, C_casted, ldc);
  }

  void cublasZtrmm_elpa_wrapper (char side, char uplo, char transa, char diag,
                               int m, int n, double complex alpha, const double complex *A,
                               int lda, double complex *B, int ldb){

    hipDoubleComplex alpha_casted = *((hipDoubleComplex*)(&alpha));
    
    const hipDoubleComplex* A_casted = (const hipDoubleComplex*) A;
    hipDoubleComplex* B_casted = (hipDoubleComplex*) B;    
    
    hipblasZtrmm(side, uplo, transa, diag, m, n, alpha_casted, A_casted, lda, B_casted, ldb);
  }

  void cublasCtrmm_elpa_wrapper (char side, char uplo, char transa, char diag,
                               int m, int n, float complex alpha, const float complex *A,
                               int lda, float complex *B, int ldb){

    hipFloatComplex alpha_casted = *((hipFloatComplex*)(&alpha));
    
    const hipFloatComplex* A_casted = (const hipFloatComplex*) A;
    hipFloatComplex* B_casted = (hipFloatComplex*) B;    
    
    hipblasCtrmm(side, uplo, transa, diag, m, n, alpha_casted, A_casted, lda, B_casted, ldb);
  }

  
}
#endif /* WITH_GPU_VERSION */
